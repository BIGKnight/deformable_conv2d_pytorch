#include "hip/hip_runtime.h"
//#include "deformable_conv2d.h"
// beware that the *cuh can only be referenced in the .cu or .cuh files, I put these two header into the .h file at first and it spend me a lot time to find this bug
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>

#include <THC/THC.h>
#include <THC/THCAtomics.cuh>
#include <THC/THCDeviceUtils.cuh>
#include <vector>

typedef std::vector<int> TShape;

inline int ProdShape(const TShape &shape, int start, int end) {
    int res = 1;
    for(int i=start; i<end; i++) {
        res*=shape[i];
    }
    return res;
}

inline TShape SubVector(const TShape &shape, int start, int end) {
    TShape res;
    for(int i=start;i<end;i++){
        res.push_back(shape[i]);
    }
    return res;
}

#define CUDA_KERNEL_LOOP(i, n)                          \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x;   \
      i < (n);                                          \
      i += blockDim.x * gridDim.x)

const int CUDA_NUM_THREADS = 1024;

inline int GET_BLOCKS(const int N)
{
  return (N + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS;
}

__device__ float dmcn_im2col_bilinear(
    const float* bottom_data,
    const int data_width,
    const int height,
    const int width,
    float h,
    float w){

  int h_low = floor(h);
  int w_low = floor(w);
  int h_high = h_low + 1;
  int w_high = w_low + 1;

  float lh = h - h_low;
  float lw = w - w_low;
  float hh = 1 - lh, hw = 1 - lw;

  float v1 = 0;
  if (h_low >= 0 && w_low >= 0)
    v1 = bottom_data[h_low * data_width + w_low];
  float v2 = 0;
  if (h_low >=0 && w_high <= width - 1)
    v2 = bottom_data[h_low * data_width + w_high];
  float v3 = 0;
  if (h_high <= height - 1 && w_low >= 0)
    v3 = bottom_data[h_high * data_width + w_low];
  float v4 = 0;
  if (h_high <= height - 1 && w_high <= width - 1)
    v4 = bottom_data[h_high * data_width + w_high];

  float w1 = hh * hw, w2 = hh * lw, w3 = lh * hw, w4 = lh * lw;

  float val = (w1 * v1 + w2 * v2 + w3 * v3 + w4 * v4);
  return val;

}

__device__ float dmcn_get_gradient_weight(
    float argmax_h, // offset h
    float argmax_w, // offset w
    const int h,  const int w, // coordinate
    const int height,  const int width){

  if (argmax_h <= -1 || argmax_h >= height || argmax_w <= -1 || argmax_w >= width) {
    //empty
    return 0;
  }

  int argmax_h_low = floor(argmax_h);
  int argmax_w_low = floor(argmax_w);
  int argmax_h_high = argmax_h_low + 1;
  int argmax_w_high = argmax_w_low + 1;

  float weight = 0;
  if (h == argmax_h_low && w == argmax_w_low)
      weight = (h + 1 - argmax_h) * (w + 1 - argmax_w);
  if (h == argmax_h_low && w == argmax_w_high)
      weight = (h + 1 - argmax_h) * (argmax_w + 1 - w);
  if (h == argmax_h_high && w == argmax_w_low)
      weight = (argmax_h + 1 - h) * (w + 1 - argmax_w);
  if (h == argmax_h_high && w == argmax_w_high)
      weight = (argmax_h + 1 - h) * (argmax_w + 1 - w);
  return weight;
}

__device__ float dmcn_get_coordinate_weight(
    float argmax_h,
    float argmax_w,
    const int height,
    const int width,
    const float* im_data,
    const int data_width,
    const int bp_dir
    ) {

  if (argmax_h <= -1 || argmax_h >= height || argmax_w <= -1 || argmax_w >= width)
  {
    //empty
    return 0;
  }

  int argmax_h_low = floor(argmax_h);
  int argmax_w_low = floor(argmax_w);
  int argmax_h_high = argmax_h_low + 1;
  int argmax_w_high = argmax_w_low + 1;

  float weight = 0;

  if (bp_dir == 0) {
    if (argmax_h_low >= 0 && argmax_w_low >= 0)
        weight += -1 * (argmax_w_low + 1 - argmax_w) * im_data[argmax_h_low * data_width + argmax_w_low];
    if (argmax_h_low >= 0 && argmax_w_high <= width - 1)
        weight += -1 * (argmax_w - argmax_w_low) * im_data[argmax_h_low * data_width + argmax_w_high];
    if (argmax_h_high <= height - 1 && argmax_w_low >= 0)
        weight += (argmax_w_low + 1 - argmax_w) * im_data[argmax_h_high * data_width + argmax_w_low];
    if (argmax_h_high <= height - 1 && argmax_w_high <= width - 1)
        weight += (argmax_w - argmax_w_low) * im_data[argmax_h_high * data_width + argmax_w_high];
  } else if (bp_dir == 1) {
    if (argmax_h_low >= 0 && argmax_w_low >= 0)
        weight += -1 * (argmax_h_low + 1 - argmax_h) * im_data[argmax_h_low * data_width + argmax_w_low];
    if (argmax_h_low >= 0 && argmax_w_high <= width - 1)
        weight += (argmax_h_low + 1 - argmax_h) * im_data[argmax_h_low * data_width + argmax_w_high];
    if (argmax_h_high <= height - 1 && argmax_w_low >= 0)
        weight += -1 * (argmax_h - argmax_h_low) * im_data[argmax_h_high * data_width + argmax_w_low];
    if (argmax_h_high <= height - 1 && argmax_w_high <= width - 1)
        weight += (argmax_h - argmax_h_low) * im_data[argmax_h_high * data_width + argmax_w_high];
  }

  return weight;
}

__global__ void SwapAxisKernel(
    const int n,
    const int cuda_mem_size, const int min_unit_size,
    float* input_data,
    const int dim_num,
    const int axis_x_dims, const int axis_y_dims,
    const int axis_x, const int axis_y){
    CUDA_KERNEL_LOOP(index, n){
//        size_t size = cuda_mem_size * sizeof(float);
        float *device_data = NULL;

        device_data = new float[cuda_mem_size];

//        hipMalloc((void**)&device_data, size);
        float* input_data_ptr = input_data + index * cuda_mem_size;
        for(int j =0;j<axis_y_dims;j++){
            for(int i=0;i<axis_x_dims;i++){
                float* temp_ptr = input_data_ptr + (i * axis_x_dims + j) * min_unit_size;
//                hipMemcpy(device_data + (j * axis_y_dims + i) * min_unit_size, temp_ptr, sizeof(float)*min_unit_size, hipMemcpyHostToDevice);
                float* device_data_temp_ptr = device_data +  (j * axis_y_dims + i) * min_unit_size;
                for(int k = 0;k<min_unit_size;k++){
                    *(device_data_temp_ptr + k) = *(temp_ptr + k);
                }
            }
        }
//        hipMemcpy(input_data_ptr, device_data, size, hipMemcpyDeviceToHost);
        for(int i =0;i<cuda_mem_size;i++)
            *(input_data_ptr + i) = *(device_data + i);
    }
}

__global__ void DeformableConv2DIm2ColKernel(
    const int n,
    const float* data_im,
    const float* data_offset,
    const float* data_mask,

    const int height, const int width,
    const int kernel_h, const int kernel_w,
    const int pad_h, const int pad_w,
    const int stride_h, const int stride_w,
    const int dilation_h, const int dilation_w,

    const int channel_per_deformable_group,
    const int batch_size, const int num_channels, const int deformable_group,
    const int height_col, const int width_col,
    float* data_col){
    CUDA_KERNEL_LOOP(index, n) {
    const int w_col = index % width_col;
    const int h_col = (index / width_col) % height_col;
    const int b_col = (index / width_col / height_col) % batch_size;
    const int c_im = (index / width_col / height_col) / batch_size;
    const int c_col = c_im * kernel_h * kernel_w;

    const int deformable_group_index = c_im / channel_per_deformable_group;

    const int h_in = h_col * stride_h - pad_h;
    const int w_in = w_col * stride_w - pad_w;

    float* data_col_ptr = data_col + ((c_col * batch_size + b_col) * height_col + h_col) * width_col + w_col;
    const float* data_im_ptr = data_im + (b_col * num_channels + c_im) * height * width;
    const float* data_offset_ptr = data_offset + (b_col * deformable_group + deformable_group_index) * 2 * kernel_h * kernel_w * height_col * width_col;

    const float* data_mask_ptr = data_mask + (b_col *  deformable_group + deformable_group_index) * kernel_h * kernel_w * height_col * width_col;

    for (int i = 0; i < kernel_h; ++i) {
      for (int j = 0; j < kernel_w; ++j) {
        const int data_offset_h_ptr = ((2 * (i * kernel_w + j)) * height_col + h_col) * width_col + w_col;
        const int data_offset_w_ptr = ((2 * (i * kernel_w + j) + 1) * height_col + h_col) * width_col + w_col;
        const int data_mask_hw_ptr = ((i * kernel_w + j) * height_col + h_col) * width_col + w_col;
        const float offset_h = data_offset_ptr[data_offset_h_ptr];
        const float offset_w = data_offset_ptr[data_offset_w_ptr];
        const float mask = data_mask_ptr[data_mask_hw_ptr];
        float val = static_cast<float>(0);
        const float h_im = h_in + i * dilation_h + offset_h;
        const float w_im = w_in + j * dilation_w + offset_w;
        if (h_im > -1 && w_im > -1 && h_im < height && w_im < width) {
          val = dmcn_im2col_bilinear(data_im_ptr, width, height, width, h_im, w_im);
        }
        *data_col_ptr = val * mask;
        data_col_ptr += batch_size * height_col * width_col;
      }
    }
  }
}

__global__ void DeformableConv2DCol2ImKernel(
    const int n,
    const float* data_col, const float* data_offset, const float* data_mask,
    const int channels, const int height, const int width,
    const int kernel_h, const int kernel_w,
    const int pad_h, const int pad_w,
    const int stride_h, const int stride_w,
    const int dilation_h, const int dilation_w,
    const int channel_per_deformable_group,
    const int batch_size, const int deformable_group,
    const int height_col, const int width_col,
    float* grad_im){
    CUDA_KERNEL_LOOP(index, n){
        const int j = (index / width_col / height_col / batch_size) % kernel_w;
        const int i = (index / width_col / height_col / batch_size / kernel_w) % kernel_h;
        const int c = index / width_col / height_col / batch_size / kernel_w / kernel_h;
        const int deformable_group_index = c / channel_per_deformable_group;
        int w_out = index % width_col;
        int h_out = (index / width_col) % height_col;
        int b = (index / width_col / height_col) % batch_size;
        int w_in = w_out * stride_w - pad_w;
        int h_in = h_out * stride_h - pad_h;
        const float* data_offset_ptr = data_offset + (b * deformable_group + deformable_group_index) * 2 * kernel_h * kernel_w * height_col * width_col;
        const float* data_mask_ptr = data_mask + (b * deformable_group + deformable_group_index) * kernel_h * kernel_w * height_col * width_col;
        const int data_offset_h_ptr = ((2 * (i * kernel_w + j)) * height_col + h_out) * width_col + w_out;
        const int data_offset_w_ptr = ((2 * (i * kernel_w + j) + 1) * height_col + h_out) * width_col + w_out;
        const int data_mask_hw_ptr = ((i * kernel_w + j) * height_col + h_out) * width_col + w_out;
        const float offset_h = data_offset_ptr[data_offset_h_ptr];
        const float offset_w = data_offset_ptr[data_offset_w_ptr];
        const float mask = data_mask_ptr[data_mask_hw_ptr];
        const float cur_inv_h_data = h_in + i * dilation_h + offset_h;
        const float cur_inv_w_data = w_in + j * dilation_w + offset_w;
        const float cur_top_grad = data_col[index] * mask;
        const int cur_h = (int)cur_inv_h_data;
        const int cur_w = (int)cur_inv_w_data;
        for (int dy = -2; dy <= 2; dy++) {
        for (int dx = -2; dx <= 2; dx++) {
            if (cur_h + dy >= 0 && cur_h + dy < height &&
            cur_w + dx >= 0 && cur_w + dx < width &&
            abs(cur_inv_h_data - (cur_h + dy)) < 1 &&
            abs(cur_inv_w_data - (cur_w + dx)) < 1
            ) {
                int cur_bottom_grad_pos = ((b * channels + c) * height + cur_h + dy) * width + cur_w + dx;
                float weight = dmcn_get_gradient_weight(cur_inv_h_data, cur_inv_w_data, cur_h + dy, cur_w + dx, height, width);
                atomicAdd(grad_im + cur_bottom_grad_pos, weight * cur_top_grad);
                }
            }
        }
    }
}

__global__ void DeformableConv2DCol2ImCoordGPUKernel(
  const int n,
  const float* data_col, const float* data_im,
  const float* data_offset, const float* data_mask,
  const int channels, const int height, const int width, // 输入的C, H, W
  const int kernel_h, const int kernel_w,
  const int pad_h, const int pad_w,
  const int stride_h, const int stride_w,
  const int dilation_h, const int dilation_w,
  const int channel_per_deformable_group,
  const int batch_size, const int offset_channels, const int deformable_group,
  const int height_col, const int width_col,
  float* grad_offset, float* grad_mask) {
  CUDA_KERNEL_LOOP(index, n){
    float val = 0, mval = 0;
    int w = index % width_col;
    int h = (index / width_col) % height_col;
    int c = (index / width_col / height_col) % offset_channels;
    int b = (index / width_col / height_col) / offset_channels;
    const int deformable_group_index = c / (2 * kernel_h * kernel_w);
    const int col_step = kernel_h * kernel_w;
    int cnt = 0;
    const float* data_col_ptr = data_col + deformable_group_index * channel_per_deformable_group * batch_size * width_col * height_col;
    const float* data_im_ptr = data_im + (b * deformable_group + deformable_group_index) * channel_per_deformable_group / kernel_h / kernel_w * height * width;
    const float* data_offset_ptr = data_offset + (b * deformable_group + deformable_group_index) * 2 * kernel_h * kernel_w * height_col * width_col;
    const float* data_mask_ptr = data_mask + (b * deformable_group + deformable_group_index) * kernel_h * kernel_w * height_col * width_col;
    const int offset_c = c - deformable_group_index * 2 * kernel_h * kernel_w;
    for (int col_c = (offset_c / 2); col_c < channel_per_deformable_group; col_c += col_step) {
      const int col_pos = (((col_c * batch_size + b) * height_col) + h) * width_col + w;
      const int bp_dir = offset_c % 2;
      int j = (col_pos / width_col / height_col / batch_size) % kernel_w;
      int i = (col_pos / width_col / height_col / batch_size / kernel_w) % kernel_h;
      int w_out = col_pos % width_col;
      int h_out = (col_pos / width_col) % height_col;
      int w_in = w_out * stride_w - pad_w;
      int h_in = h_out * stride_h - pad_h;
      const int data_offset_h_ptr = (((2 * (i * kernel_w + j)) * height_col + h_out) * width_col + w_out);
      const int data_offset_w_ptr = (((2 * (i * kernel_w + j) + 1) * height_col + h_out) * width_col + w_out);
      const int data_mask_hw_ptr = (((i * kernel_w + j) * height_col + h_out) * width_col + w_out);
      const float offset_h = data_offset_ptr[data_offset_h_ptr];
      const float offset_w = data_offset_ptr[data_offset_w_ptr];
      const float mask = data_mask_ptr[data_mask_hw_ptr];
      float inv_h = h_in + i * dilation_h + offset_h;
      float inv_w = w_in + j * dilation_w + offset_w;
      if (inv_h <= -1 || inv_w <= -1 || inv_h >= height || inv_w >= width) {
        inv_h = inv_w = -2;
      } else {
        mval += data_col_ptr[col_pos] * dmcn_im2col_bilinear(data_im_ptr + cnt * height * width, width, height, width, inv_h, inv_w);
      }
      const float weight = dmcn_get_coordinate_weight(
        inv_h, inv_w,
        height, width, data_im_ptr + cnt * height * width, width, bp_dir);
      val  += weight * data_col_ptr[col_pos] * mask;
      cnt  += 1;
    }

    grad_offset[index] = val;
    if (offset_c % 2 == 0){
            grad_mask[(((b * deformable_group + deformable_group_index) * kernel_h * kernel_w + offset_c / 2) * height_col + h) * width_col + w] = mval;
        }
    }
}

__global__ void pureAddToKernel(const int n, float* result_data, const float* right_data){
      CUDA_KERNEL_LOOP(index, n) {
          atomicAdd(result_data+index, right_data[index]);
      }
    }

__global__ void setZeroKernel(const int n, float* result_data){
         CUDA_KERNEL_LOOP(index, n){
          *(result_data + index) = float(0);
      }

    }

__global__ void setOneKernel(const int n, float* result_data){
        CUDA_KERNEL_LOOP(index, n){
            *(result_data + index) = float(1);
        }
    }

void deformable_im2col(hipStream_t stream,
    const float* data_im, const float* data_offset, const float* data_mask,
    const TShape& im_shape, const TShape& col_shape, const TShape& kernel_shape,
    const TShape& pad, const TShape& stride, const TShape& dilation,
    const int deformable_group, float* data_col) {
        int  num_spatial_axes = kernel_shape.size();
        int  channel_per_deformable_group = im_shape[1] / deformable_group;
        int  num_kernels = im_shape[1] * ProdShape(col_shape, 1, col_shape.size());
        switch (num_spatial_axes) {
        case 2:
        DeformableConv2DIm2ColKernel // NOLINT_NEXT_LINE(whitespace/operators)
            <<<GET_BLOCKS(num_kernels), CUDA_NUM_THREADS, 0, stream>>>(
            num_kernels,
            data_im,
            data_offset,
            data_mask,
            im_shape[2], im_shape[3],
            kernel_shape[0], kernel_shape[1],
            pad[0], pad[1],
            stride[0], stride[1],
            dilation[0], dilation[1],
            channel_per_deformable_group,
            col_shape[1], im_shape[1],
            deformable_group,
            col_shape[2], col_shape[3],
            data_col);
            break;
            default:
                hipError_t err = hipGetLastError();
                printf("error in DeformableConv2DIm2ColKernel: %s\n", hipGetErrorString(err));
            }

}

void deformable_col2im(hipStream_t stream,
    const float* data_col, const float* data_offset, const float* data_mask,
    const TShape& im_shape, const TShape& col_shape, const TShape& kernel_shape,
    const TShape& pad, const TShape& stride,
    const TShape& dilation, const int deformable_group,
    float* grad_im){
        int  num_spatial_axes = kernel_shape.size();
        int  im_size = ProdShape(im_shape, 1, im_shape.size());
        int  channel_per_deformable_group = im_shape[1] / deformable_group;
        int  num_kernels = ProdShape(col_shape, 0, col_shape.size());
          switch (num_spatial_axes) {
          case 2:
                DeformableConv2DCol2ImKernel
                <<<GET_BLOCKS(num_kernels), CUDA_NUM_THREADS, 0, stream>>>(
                num_kernels, data_col, data_offset, data_mask, im_shape[1], im_shape[2], im_shape[3],
                kernel_shape[0], kernel_shape[1], pad[0], pad[1], stride[0], stride[1],
                dilation[0], dilation[1], channel_per_deformable_group,
                col_shape[1], deformable_group, col_shape[2], col_shape[3], grad_im);
            break;
          default:
            hipError_t err = hipGetLastError();
            printf("error in DeformableConv2DIm2ColKernel: %s\n", hipGetErrorString(err));
          }

}



void deformable_col2im_coord(hipStream_t stream,
    const float* data_col, const float* data_im, const float* data_offset, const float* data_mask,
    const TShape& im_shape, const TShape& col_shape, const TShape& kernel_shape,
    const TShape& pad, const TShape& stride,
    const TShape& dilation, const int deformable_group,
    float* grad_offset, float* grad_mask) {
      int  num_spatial_axes = kernel_shape.size();
      int  num_kernels = col_shape[1] * col_shape[2] * col_shape[3] * 2 * kernel_shape[0] * kernel_shape[1] * deformable_group;
      int  channel_per_deformable_group = col_shape[0] / deformable_group;
      switch (num_spatial_axes) {
      case 2:
        DeformableConv2DCol2ImCoordGPUKernel
        <<<GET_BLOCKS(num_kernels), CUDA_NUM_THREADS, 0, stream>>>(
            num_kernels, data_col, data_im, data_offset, data_mask, im_shape[1], im_shape[2], im_shape[3],
            kernel_shape[0], kernel_shape[1], pad[0], pad[1], stride[0], stride[1],
            dilation[0], dilation[1], channel_per_deformable_group,
            col_shape[1], 2 * kernel_shape[0] * kernel_shape[1] * deformable_group, deformable_group, col_shape[2], col_shape[3],
            grad_offset, grad_mask);
        break;
      default:
            hipError_t err = hipGetLastError();
            printf("error in DeformableConv2DCol2ImCoordGPUKernel: %s\n", hipGetErrorString(err));
    }
}

void SwapAxis(hipStream_t stream, float* input_data, const TShape& origin_shape, const int axis_x, const int axis_y){
    return;
}

void setZero(hipStream_t stream, int n, float* result_data){
    setZeroKernel <<< GET_BLOCKS(n), CUDA_NUM_THREADS, 0, stream >>>(n, result_data);
}

void setOne(hipStream_t stream, int n, float* result_data){
    setOneKernel <<< GET_BLOCKS(n), CUDA_NUM_THREADS, 0, stream >>>(n, result_data);
}

void pureAddTo(hipStream_t stream, const int n, float* result_data, const float* right_data){
    pureAddToKernel<<< GET_BLOCKS(n), CUDA_NUM_THREADS, 0, stream  >>>(n, result_data, right_data);
}

void setNumAtIndex(hipStream_t stream,  float num, int index, float* data){
//     *(data + index) = num;
}



